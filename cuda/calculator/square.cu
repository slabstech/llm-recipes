
#include <hip/hip_runtime.h>
#include <stdio.h>

// kernel to be run on the TX2
__global__ void square(float *d_out, float *d_in){
  int idx = threadIdx.x; // this is how you get the thread index
  float f = d_in[idx];
  d_out[idx] = f*f;
}

// main is here. this is the CPU code. 
int main(){
  // the size of the array, which is really the thread count per block
  const int ARRAY_SIZE = 10; // max thread count per block on the TX2
  // total bytes in the array
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  // generate the input array on the host for the device my dude
  float h_in[ARRAY_SIZE];
  for (int i = 0; i < ARRAY_SIZE; i++){
    h_in[i] = float(i);
  }
  float h_out[ARRAY_SIZE]; // make the other array too my guy
  
  // make the pointers that point to the mem locations on the GPU m'lady
  float *d_in;
  float *d_out;

  //allocate the memory on the GPU my man
  hipMalloc((void **) &d_in, ARRAY_BYTES);
  hipMalloc((void **) &d_out, ARRAY_BYTES);

  // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

  // launch the kernel
  square<<<1,ARRAY_SIZE>>>(d_out,d_in);

  // copy the result back to the CPU mem
  hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

  // print the result
  for (int i = 0; i < ARRAY_SIZE; i++){
    printf("%f", h_out[i]);
    printf(((i % 4) != 3) ? "\t" : "\n");
  }

  // free the device yo
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}
